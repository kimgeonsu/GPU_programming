#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    const int n = 5;
    const int rows = n;

    double host_A[25] = {5,2,3,1, -10,
                         7,3,2,-4,1,
                         5,3,9,1,2,
                         2,1,-2,8,2,
                         9,3,1,-9,3
                        };
    double host_b[5] = {10, 2, 3, -4, 5};
    double host_x[5] = {0, 0, 0, 0, 0};

    double *A, *b, *x, *r, *p, *Axp;
    hipMalloc((void**)&A, n * n * sizeof(double));
    hipMalloc((void**)&b, n * sizeof(double));
    hipMalloc((void**)&x, n * sizeof(double));
    hipMalloc((void**)&r, n * sizeof(double));
    hipMalloc((void**)&p, n * sizeof(double));
    hipMalloc((void**)&Axp, n * sizeof(double));
    
    hipMemcpy(A, host_A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b, host_b, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x, host_x, n * sizeof(double), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double zero = 0.0, one = 1.0, minusOne = -1.0;
    double alpha = 0.0, beta = 0.0, rxr = 0.0, tmp = 0.0;
    double epsilon = 1e-6;
    int maxit = 100000000;

    hipblasDcopy(handle, n, b, 1, r, 1);

    hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &minusOne, A, rows, x, 1, &one, r, 1);

    hipblasDcopy(handle, n, r, 1, p, 1);
    hipblasDdot(handle, n, r, 1, r, 1, &rxr);

    int k = 0;
    while (k < maxit) {
        hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &one, A, rows, p, 1, &zero, Axp, 1); // Axp = A * p
        hipblasDdot(handle, n, p, 1, Axp, 1, &tmp);
        alpha = rxr / tmp;

        hipblasDaxpy(handle, n, &alpha, p, 1, x, 1);
        tmp = -alpha;
        hipblasDaxpy(handle, n, &tmp, Axp, 1, r, 1); // r = r - alpha * Axp
        hipblasDdot(handle, n, r, 1, r, 1, &tmp);

        if (sqrt(tmp) < epsilon) {
            break;
        }

        beta = tmp / rxr;
        rxr = tmp;
        hipblasDscal(handle, n, &beta, p, 1); // p = beta * p
        hipblasDaxpy(handle, n, &one, r, 1, p, 1);

        k++;
    }
    hipMemcpy(host_x, x, n * sizeof(double), hipMemcpyDeviceToHost);
    
    int i = 0;
    for (i = 0; i < n; i++) {
        printf("x[%d] = %f\n", i, host_x[i]);
    }

    hipFree(A);
    hipFree(b);
    hipFree(x);
    hipFree(r);
    hipFree(p);
    hipFree(Axp);
    hipblasDestroy(handle);
    return 0;
}