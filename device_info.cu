
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void myKernelHello(void){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    printf("[%d, %d] - thread-ID:%d\n", blockIdx.x, threadIdx.x, tid);
}

int main() {
    hipDeviceProp_t props;
    printf("\tdevice\n");
    hipGetDeviceProperties(&props, 0);
    printf("\t\tname: %s\n", props.name);

    return 0;
}