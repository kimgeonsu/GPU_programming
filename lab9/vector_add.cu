
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void initWith(float num, float* a, const int N) {
    for (int i = 0; i < N; i++) {
        a[i] = num;
    }
}

__global__
void addVectorsInto(float* ressult, float* a, float* b, const int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride) {
        ressult[i] = a[i] + b[i];
    }
}

void checkElmentAre(float target, float* array, const int N) {
    for (int i = 0; i < N; i++) {
        if (array[i] != target) {
            printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
            exit(1);
        }
    }
    printf("SUCCESS! All values added correctly.\n");
}

int main() {
    const int N = 2 << 20;
    size_t size = N * sizeof(float);

    float *a, *b, *c;

    checkCuda(hipMallocManaged(&a, size));
    checkCuda(hipMallocManaged(&b, size));
    checkCuda(hipMallocManaged(&c, size));

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    size_t threadsPerBlock = 1;
    size_t numberOfBlocks = 1;
    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    checkElmentAre(7, c, N);

    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(c));
}